#include "hip/hip_runtime.h"
﻿#include "cudaRun.h"

#define IDX2C(i,j,ld) (((j)*(ld))+(i))

void cudaCompute()
{
	clock_t t_begin, t_end;
	t_begin = clock();
	// ====================================

	hipblasStatus_t stat;
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	// var on the host
	float* h_in; // 存储新的结果
	float* h_in_A; // 被加数，不变
	float* h_out;
	float* h_in_B;

	h_in = (float*)malloc(SIZE*SIZE * sizeof(float));
	h_in_A = (float*)malloc(SIZE*SIZE * sizeof(float));
	h_out = (float*)malloc(SIZE*SIZE * sizeof(float));
	h_in_B = (float*)malloc(SIZE*SIZE * sizeof(float));
	for (int i = 0; i < SIZE*SIZE; i++)
	{
		h_in[i] = (float)(1);
		h_in_A[i] = (float)(1);
	}

	// B 的主对角线上是1， 其余是0
	for (int i = 0; i < SIZE; i++)
	{
		for (int j = 0; j < SIZE; j++)
		{
			if (i==j)
			{
				h_in_B[IDX2C(i,j,SIZE)] = (float)(1);
			}
			else
			{
				h_in_B[IDX2C(i,j,SIZE)] = (float)(0);
			}
		}
	}

	// var on the device
	float* d_in;
	float* d_in_A;
	float* d_in_B;
	hipMalloc((void**)&d_in, SIZE*SIZE * sizeof(float));
	hipMalloc((void**)&d_in_A, SIZE*SIZE * sizeof(float));
	hipMalloc((void**)&d_in_B, SIZE*SIZE * sizeof(float));
	hipMemcpy(d_in, h_in, SIZE*SIZE * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_in_A, h_in_A, SIZE*SIZE * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_in_B, h_in_B, SIZE*SIZE * sizeof(float), hipMemcpyHostToDevice);
	// cublas level 3
	// C = alpha * A * B + beta * C
	float alpha = 1.0f;
	float beta = 1.0f;

	for (int i = 0; i < loops; i++)
	{
		hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
			SIZE, SIZE, SIZE,
			&alpha,
			d_in_A, SIZE,
			d_in_B, SIZE,
			&beta,
			d_in, SIZE);
	}

	hipMemcpy(h_out, d_in, SIZE*SIZE * sizeof(float), hipMemcpyDeviceToHost);

	// 展示前10个数
	for (int i = 0; i < 10; i++)
	{
		printf("%f\n",h_out[i]);
	}

	// free
	free(h_in);
	free(h_out);
	free(h_in_B);
	hipFree(d_in);
	hipFree(d_in_B);
	hipblasDestroy(handle);
	// ====================================
	t_end = clock();
	printf("---------------------------\n");
	printf("cuda, time needed: %f s\n", 
		(double)(t_end-t_begin)/CLOCKS_PER_SEC);
	printf("============================\n");
}
