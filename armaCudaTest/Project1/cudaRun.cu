#include "hip/hip_runtime.h"
﻿#include "cudaRun.h"

#define IDX2C(i,j,ld) (((j)*(ld))+(i))

void cudaComputeAdd()
{
	clock_t t_begin, t_end;
	t_begin = clock();
	// ====================================

	hipblasStatus_t stat;
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	// var on the host
	float* h_in; // 存储新的结果
	float* h_in_A; // 被加数，不变
	float* h_out;
	float* h_in_B;

	h_in = (float*)malloc(SIZE*SIZE * sizeof(float));
	h_in_A = (float*)malloc(SIZE*SIZE * sizeof(float));
	h_out = (float*)malloc(SIZE*SIZE * sizeof(float));
	h_in_B = (float*)malloc(SIZE*SIZE * sizeof(float));
	for (int i = 0; i < SIZE*SIZE; i++)
	{
		h_in[i] = (float)(1);
		h_in_A[i] = (float)(1);
	}

	// B 的主对角线上是1， 其余是0
	for (int i = 0; i < SIZE; i++)
	{
		for (int j = 0; j < SIZE; j++)
		{
			if (i==j)
			{
				h_in_B[IDX2C(i,j,SIZE)] = (float)(1);
			}
			else
			{
				h_in_B[IDX2C(i,j,SIZE)] = (float)(0);
			}
		}
	}

	// var on the device
	// 本质上A B都是一维的，所以可以使用level 1 中的指令
	float* d_in;
	float* d_in_A;
	float* d_in_B;
	hipMalloc((void**)&d_in, SIZE*SIZE * sizeof(float));
	hipMalloc((void**)&d_in_A, SIZE*SIZE * sizeof(float));
	hipMalloc((void**)&d_in_B, SIZE*SIZE * sizeof(float));
	hipMemcpy(d_in, h_in, SIZE*SIZE * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_in_A, h_in_A, SIZE*SIZE * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_in_B, h_in_B, SIZE*SIZE * sizeof(float), hipMemcpyHostToDevice);
	// cublas level 3
	// C = alpha * A * B + beta * C 会把计算结果放到 C 中
	float alpha = 1.0f;
	float beta = 1.0f;

	for (int i = 0; i < LOOPS; i++)
	{
		hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
			SIZE, SIZE, SIZE,
			&alpha,
			d_in_A, SIZE,
			d_in_B, SIZE,
			&beta,
			d_in, SIZE);
	}

	hipMemcpy(h_out, d_in, SIZE*SIZE * sizeof(float), hipMemcpyDeviceToHost);

	// 展示前10个数
	for (int i = 0; i < 10; i++)
	{
		printf("%f\n",h_out[i]);
	}

	// free
	free(h_in);
	free(h_out);
	free(h_in_B);
	hipFree(d_in);
	hipFree(d_in_B);
	hipblasDestroy(handle);
	// ====================================
	t_end = clock();
	printf("---------------------------\n");
	printf("cuda, time needed: %f s\n", 
		(double)(t_end-t_begin)/CLOCKS_PER_SEC);
	printf("============================\n");
}

void cudaComputeMul()
{
	clock_t t_begin, t_end;
	t_begin = clock();
	// ====================================




	// ====================================
	t_end = clock();
	printf("---------------------------\n");
	printf("cuda, time needed: %f s\n",
		(double)(t_end - t_begin) / CLOCKS_PER_SEC);
	printf("============================\n");
}

void cudaFnL2()
{
	clock_t t_begin, t_end;
	t_begin = clock();
	// ====================================
	hipblasStatus_t stat;
	hipblasHandle_t handle;
	hipblasCreate(&handle);

	// 探究 level 1 2 3 的常用方法
	// ----------- level 1 --------------------
	



	// ----------- level 2 --------------------
	// gemv: y = alpha * A' * x + beta * y
	float alpha = 1, beta = 0; // y=A' * x
	
	// var on the host
	float* h_in_A;
	float* h_in_x;
	float* h_y;
	h_in_A = (float*)malloc(M*N * sizeof(float));
	h_in_x = (float*)malloc(M * sizeof(float));
	h_y = (float*)malloc(N*sizeof(float));
	int count = 0;
	for (int i = 0; i < M; i++)
	{
		for (int j = 0; j < N; j++)
		{
			h_in_A[IDX2C(i, j, M)] = count++;
		}
	}
	/*
		0, 1, 2,
		3, 4, 5,
		.
		.
		.
	*/
	/*printf("A:\n");
	printMat(h_in_A, M, N);*/

	for (int i = 0; i < M; i++)
	{
		h_in_x[i] = 1.0f;
	}
	/*printf("x:\n");
	printVec(h_in_x, M);*/

	// var on the device
	float* d_in_A;
	float* d_in_x;
	float* d_y;
	hipMalloc((void**)&d_in_A, M*N * sizeof(float));
	hipMalloc((void**)&d_in_x, M * sizeof(float));
	hipMalloc((void**)&d_y, N * sizeof(float));
	hipMemcpy(d_in_A, h_in_A, M*N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_in_x, h_in_x, M * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_y, h_y, N * sizeof(float), hipMemcpyHostToDevice);

	// compute with cuBLAS
	for (int i = 0; i < LOOPS; i++)
	{
		hipblasSgemv(handle,
			HIPBLAS_OP_T, // A'
			M, N, // n_rows, n_cols of A
			&alpha,
			d_in_A, M, // M 是dla，即leading dimension of A
			d_in_x, 1,
			&beta,
			d_y, 1); // y stores the result
	}

	hipMemcpy(h_y, d_y, N * sizeof(float), hipMemcpyDeviceToHost);
	/*printf("y:\n");
	printVec(h_y, N);*/

	// free
	free(h_in_A);
	free(h_in_x);
	free(h_y);
	hipFree(d_in_A);
	hipFree(d_in_x);
	hipFree(d_y);

	// ----------- level 3 --------------------


	hipblasDestroy(handle);

	// ====================================
	t_end = clock();
	printf("\n");
	printf("cuda, time needed: %f s\n",
		(double)(t_end - t_begin) / CLOCKS_PER_SEC);
	printf("=====================\n");
}

void h2D2H(float alpha, float * h_in_A, float * h_in_x, float beta, float * h_y)
{

}

__global__ void global_elememtMul(float * d_in_vec1, float * d_in_vec2, int size, float * d_out)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < size)
	{
		int v1 = d_in_vec1[tid];
		int v2 = d_in_vec2[tid];
		d_out[tid] = v1 * v2;
	}

}

void elementMul(float * h_in_vec1, float * h_in_vec2, int size, float * h_res)
{
	// var on the host are already init and assigned

	// var on the device
	float* d_in_vec1;
	float* d_in_vec2;
	float* d_out;
	hipMalloc((void**)&d_in_vec1, size * sizeof(float));
	hipMalloc((void**)&d_in_vec2, size * sizeof(float));
	hipMalloc((void**)&d_out, size * sizeof(float));
	hipMemcpy(d_in_vec1, h_in_vec1, size * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_in_vec2, h_in_vec2, size * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_out, h_res, size * sizeof(float), hipMemcpyHostToDevice);
	
	
	// launch kernel
	global_elememtMul << <2, 1024 >> > (d_in_vec1, d_in_vec2, size, d_out);

	// copy back to host
	hipMemcpy(h_res, d_out, size * sizeof(float), hipMemcpyDeviceToHost);

}

void elementMulRun()
{
	// var on the host
	float* h_in_vec1;
	float* h_in_vec2;
	int size = 1025;
	float* h_res;
	h_in_vec1 = (float*)malloc(size * sizeof(float));
	h_in_vec2 = (float*)malloc(size * sizeof(float));
	h_res = (float*)malloc(size * sizeof(float));

	for (int i = 0; i < size; i++)
	{
		h_in_vec1[i] = float(i);
		h_in_vec2[i] = float(1);
	}
	printf("vec1: \n");
	printVec(h_in_vec1, size);
	printf("vec2: \n");
	printVec(h_in_vec2, size);

	// ----------------- cuda fn --------------------
	elementMul(h_in_vec1, h_in_vec2, size, h_res);

	printf("elem Mul: \n");
	printVec(h_res, size);

}



void printMat(float * mat, int n_rows, int n_cols)
{
	for (int i = 0; i < n_rows; i++)
	{
		for (int j = 0; j < n_cols; j++)
		{
			printf("%f\t", mat[j*n_rows + i]);
		}
		printf("\n");
	}
}

void printVec(float * vec, int size)
{
	for (int i = 0; i < size; i++)
	{
		printf("%f\t", vec[i]);
	}
	printf("\n");
}

__global__ void vec_add(float * x, float * y, float * z, int n)
{
	int tid = get_tid();

	if (tid < n)
	{
		z[tid] += x[tid] + y[tid];
	}
}

void vec_add_host() {
	int N = 1000000;
	int nbytes = N * sizeof(float);

	/* 1D block */
	int bs = 256;

	/* 2D grid */
	int s = ceil(sqrt((N + bs - 1.) / bs));
	dim3 grid = dim3(s, s);

	float *dx = NULL, *hx = NULL;
	float *dy = NULL, *hy = NULL;
	float *dz = NULL, *hz = NULL;

	int itr = 30; // loops
	int i;
	float th, td;

	/* allocate GPU mem */
	hipMalloc((void**)&dx, nbytes);
	hipMalloc((void**)&dy, nbytes);
	hipMalloc((void**)&dz, nbytes);

	/* allocate CPU mem */
	hx = (float*)malloc(nbytes);
	hy = (float*)malloc(nbytes);
	hz = (float*)malloc(nbytes);

	/* init */
	for (int i = 0; i < N; i++)
	{
		hx[i] = 1.;
		hy[i] = 1.;
		hz[i] = 1.;
	}

	/* copy data to GPU */
	hipMemcpy(dx, hx, nbytes, hipMemcpyHostToDevice);
	hipMemcpy(dy, hy, nbytes, hipMemcpyHostToDevice);
	hipMemcpy(dz, hz, nbytes, hipMemcpyHostToDevice);

	/* call GPU */
	hipDeviceSynchronize();
	for (int i = 0; i < itr; i++)
	{
		// launch kernel on the device
		vec_add << <grid, bs >> > (dx, dy, dz, N);
	}
	hipDeviceSynchronize(); 
	// 由于kernel调用对host而言是异步的，所以使用此行 wait for device to finish
	hipMemcpy(hz, dz, nbytes, hipMemcpyDeviceToHost);
	for (int i = 0; i < 10; i++)
	{
		printf("%f\n", hz[i]);
	}

	hipFree(dx);
	hipFree(dy);
	hipFree(dz);
	free(hx);
	free(hy);
	free(hz);

}





